#include "hip/hip_runtime.h"
/*
 *  Miller-Rabin Test Kernal
 *  Darrin Weng
*/

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#include "Miller_Rabin_Test.h"

#define GRID_SIZE 1
#define BLOCK_SIZE 512
#define THREADS_PER_NUM 32

__device__ uint32_t modular_exponent_32(uint32_t base, uint32_t power, uint32_t modulus) 
{
    uint64_t result = 1;
    int i; 
    for (i = 32; i > 0; i--) 
    {
        result = (result*result) % modulus;
        if (power & (1 << i)) 
        {
            result = (result*base) % modulus;
        }
    }
    return (uint32_t)result; /* Will not truncate since modulus is a uint32_t */
}

__global__ void setup_kernel ( hiprandState *state , int seed)
{
    int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    /* Each thread gets same seed , a different sequence number  -
    ,
    no offset */
    hiprand_init (seed, id, 0, &state[id]) ;
}


__global__ void Miller_Rabin_Kernal(Test_Result *results, hiprandState *state)
{
    //int index = (threadIdx.x % THREADS_PER_NUM) + blockIdx.x * BLOCK_SIZE;
    //int test_num = results[index].num;

    //mod random number so that a < n
    results[threadIdx.x].num = hiprand(&state[threadIdx.x]) % 10000; // % test_num; //debug
    results[threadIdx.x].passed = 0;
    /*
    //do test
    uint32_t a_to_power, s, d, i;

    //16-bit compute s and d
    s = 0;
    d = test_num - 1;

    while ((d % 2) == 0) 
    {
        d /= 2;
        s++;
    }

    a_to_power = modular_exponent_32(a, d, test_num);

    if (a_to_power == 1)
        return;

    for(i=0; i < s-1; i++) 
    {
        if (a_to_power == test_num - 1) 
            return;

        a_to_power = modular_exponent_32(a_to_power, 2, test_num);
    }

    if (a_to_power == test_num - 1)
        return;

    results[index].passed++;
    */
}

int main()
{
    hiprandState *dev_rand_state;
    Test_Result *results, *dev_results;
    uint32_t *dev_test_numbers;
    uint32_t test_numbers[BLOCK_SIZE]; //debug size right now 
    
    //Generate or get from a file the test numbers
    for(int i = 0; i < BLOCK_SIZE; i++)
    {
        test_numbers[i] = i + 1000;
    }
    
    
    //Allocate mem for RNG states, numbers to be tested, and results
    hipMalloc((void **) &dev_rand_state, sizeof(hiprandState) * BLOCK_SIZE * GRID_SIZE);
    hipMalloc((void **) &dev_results, sizeof(Test_Result) * BLOCK_SIZE * GRID_SIZE);
    hipMalloc((void **) &dev_test_numbers, sizeof(uint32_t) * BLOCK_SIZE * GRID_SIZE);
    
    results = (Test_Result *) malloc(sizeof(Test_Result) * BLOCK_SIZE * GRID_SIZE);
    
    //set up grid and blocksize
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(GRID_SIZE, 1);
    
    //seed RND with cpu rand
    srand((unsigned) time(NULL));
    int seed = rand();
    printf("Seed is %d\n", seed);
    
    //Init RNG states and transfer data to GPU
    setup_kernel<<<dimGrid, dimBlock>>>(dev_rand_state, seed);
    
    hipMemcpy(dev_test_numbers, test_numbers, sizeof(uint32_t) * BLOCK_SIZE * GRID_SIZE, 
            hipMemcpyHostToDevice);
    
    //Run Tests
    Miller_Rabin_Kernal<<<dimGrid, dimBlock>>>(dev_results, dev_rand_state);
    
    //Transfer results back to cpu
    hipMemcpy(results, dev_results, sizeof(Test_Result) * BLOCK_SIZE * GRID_SIZE, 
            hipMemcpyDeviceToHost);
            
    //Clean up memory
    hipFree(dev_results);
    hipFree(dev_rand_state);
    hipFree(dev_test_numbers);
    
    //Print results
    for(int i = 0; i < BLOCK_SIZE * GRID_SIZE; i++)
    {
        printf("%u is ", results[i].num);
        
        if(results[i].passed == PASSED)
            printf("PRIME\n");
        else
            printf("COMPOSITE\n");
    }
    
    free(results);
}

