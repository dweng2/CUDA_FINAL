#include "hip/hip_runtime.h"
/*
 *  Miller-Rabin Test Kernal
 *  Darrin Weng
*/

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#include "Miller_Rabin_Test.h"

#define GRID_SIZE 1
#define BLOCK_SIZE 512
#define THREADS_PER_NUM 32

#define CUDA_CALL(x) if(x != hipSuccess)\
printf("CUDA error %s\n", hipGetErrorString(hipGetLastError()))

__device__ uint32_t modular_exponent_32(uint32_t base, uint32_t power, uint32_t modulus) 
{
    uint64_t result = 1;
    int i; 
    for (i = 32; i > 0; i--) 
    {
        result = (result*result) % modulus;
        if (power & (1 << i)) 
        {
            result = (result*base) % modulus;
        }
    }
    return (uint32_t)result; /* Will not truncate since modulus is a uint32_t */
}

__global__ void setup_kernel ( hiprandState *state , int seed)
{
    int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    /* Each thread gets same seed , a different sequence number  -
    ,
    no offset */
    hiprand_init (seed, id, 0, &state[id]) ;
}


__global__ void Miller_Rabin_Kernal(Test_Result *results, hiprandState *state)
{
    int index = (threadIdx.x / THREADS_PER_NUM) + blockIdx.x * BLOCK_SIZE;
    printf("Thread #%d index is %d\n", threadIdx.x, index); 
    int test_num = results[index].num;

    //mod random number so that a < n
    uint32_t a = hiprand(&state[threadIdx.x]) % test_num;

    results[index].passed = 1;
    return;

    //do test
    uint32_t a_to_power, s, d, i;

    //16-bit compute s and d
    s = 0;
    d = test_num - 1;

    while ((d % 2) == 0) 
    {
        d /= 2;
        s++;
    }

    a_to_power = modular_exponent_32(a, d, test_num);

    if (a_to_power == 1)
    {
        printf("Thread #%d Return 1\n", threadIdx.x);   
        return;
    }

    for(i = 0; i < s - 1; i++) 
    {
        if (a_to_power == test_num - 1)
        {
            printf("Thread #%d Return 2\n", threadIdx.x);   
            return;
        }

        a_to_power = modular_exponent_32(a_to_power, 2, test_num);
    }

    if (a_to_power == test_num - 1)
    {
        printf("Thread #%d Return 3\n", threadIdx.x);   
        return;
    }

    printf("Thread #%d %u Not prime\n", threadIdx.x, test_num);
    results[index].passed = 1;
}

int main()
{
    hiprandState *dev_rand_state;
    Test_Result *results, *dev_results;
    int num_results = (BLOCK_SIZE / THREADS_PER_NUM) * GRID_SIZE;

    //results = (Test_Result *) malloc(sizeof(Test_Result) * num_results));
    results = (Test_Result *) malloc(sizeof(Test_Result) * num_results);

    //Generate or get from a file the test numbers
    for(int i = 0; i < num_results; i++)
    {
        results[i].num = i + 1000;
        results[i].passed = 0;
    }

    //Allocate mem for RNG states, numbers to be tested, and results
    hipMalloc((void **) &dev_rand_state, sizeof(hiprandState) * BLOCK_SIZE * GRID_SIZE);
    hipMalloc((void **) &dev_results, sizeof(Test_Result) * num_results);
    
    //set up grid and blocksize
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(GRID_SIZE, 1);
    
    //seed RND with cpu rand
    srand((unsigned) time(NULL));
    int seed = rand();
    
    //Init RNG states and transfer data to GPU
    setup_kernel<<<dimGrid, dimBlock>>>(dev_rand_state, seed);
    
    hipMemcpy(dev_results, results, sizeof(Test_Result) * num_results, hipMemcpyHostToDevice);
    
    //Run Tests
    Miller_Rabin_Kernal<<<dimGrid, dimBlock>>>(dev_results, dev_rand_state);
    
    //Transfer results back to cpu
    hipMemcpy(results, dev_results, sizeof(Test_Result) * num_results, hipMemcpyDeviceToHost);
            
    //Clean up memory
    hipFree(dev_results);
    hipFree(dev_rand_state);
    
    //Print results
    for(int i = 0; i < num_results; i++)
    {
        printf("%u is %d\n", results[i].num, results[i].passed);
        
        /*if(results[i].passed == PASSED)
            printf("PRIME\n");
        else
            printf("COMPOSITE\n");
        */
    }
    
    free(results);
}

