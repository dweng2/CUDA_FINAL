#include "hip/hip_runtime.h"
/*
 *  Miller-Rabin Test Kernal
 *  Darrin Weng
*/

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#include "Miller_Rabin_Test.h"

#define GRID_SIZE 100 //DONT MAKE TOO BIG WILL KILL MACHINES
#define BLOCK_SIZE 512
#define THREADS_PER_NUM 32
#define KERNEL_SIZE  ((BLOCK_SIZE / THREADS_PER_NUM) * GRID_SIZE)

#define CUDA_ERROR()  printf("%s\n", hipGetErrorString(hipGetLastError()));

__device__ uint32_t modular_exponent_32(uint32_t base, uint32_t power, uint32_t modulus) 
{
    uint64_t result = 1;
    
    while(power > 0)
    {
        if((power & 1) == 1)
        {
            result = (result * base) % modulus;
        }
        
        power >>= 1;
        uint64_t temp = (uint64_t) base * base;
        base = temp % modulus;
    }
    
    return (uint32_t) result; /* Will not truncate since modulus is a uint32_t */
}

__global__ void setup_kernel ( hiprandState *state , int seed)
{
    int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;

    hiprand_init (seed, id, 0, &state[id]) ;
}

__global__ void Miller_Rabin_Kernal(Test_Result *results, hiprandState *state)
{
    int index = (threadIdx.x / THREADS_PER_NUM) + blockIdx.x * (BLOCK_SIZE / THREADS_PER_NUM);
    uint32_t test_num = results[index].num;
    
    //mod random number so that a < n
    uint32_t a = 0;
    while(a < 1 || a > test_num - 1)
        a = hiprand(&state[threadIdx.x]) % test_num;

    //do test
    uint32_t a_to_power, s, d, i;

    //16-bit compute s and d
    s = 0;
    d = test_num - 1;

    while ((d % 2) == 0) 
    {
        d >>= 1;
        s++;
    }
    
    if(s == 0) //Even number so cannot be prime
    {
        results[index].passed = 1;
        return;
    }

    a_to_power = modular_exponent_32(a, d, test_num);
    
    if (a_to_power == 1)
    {
        //printf("Thread #%d %u Return 1\n", threadIdx.x, test_num);
        return;
    }

    for(i = 0; i < s - 1; i++) 
    {
        if (a_to_power == test_num - 1)
        {
            //printf("Thread #%d %u Return 2\n", threadIdx.x, test_num);
            return;
        }

        a_to_power = modular_exponent_32(a_to_power, 2, test_num);
    }

    if (a_to_power == test_num - 1)
    {
        //printf("Thread #%d %u Return 3\n", threadIdx.x, test_num);
        return;
    }
    
    //printf("Thread #%d %u Return NOT\n", threadIdx.x, test_num);
    results[index].passed = 1;
}

void run_kernel(Test_Result *results, int num_results)
{
    printf("Running CUDA\n");
    hiprandState *dev_rand_state;
    Test_Result *dev_results;
    
    //Allocate mem for RNG states, numbers to be tested, and results
    hipMalloc((void **) &dev_rand_state, sizeof(hiprandState) * BLOCK_SIZE * GRID_SIZE);
    hipMalloc((void **) &dev_results, sizeof(Test_Result) * num_results);
        
    //set up grid and blocksize
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(GRID_SIZE, 1);
    
    //seed RND with cpu rand
    srand(time(NULL));
    int seed = rand();
    
    //Init RNG states and transfer data to GPU
    setup_kernel<<<dimGrid, dimBlock>>>(dev_rand_state, seed);
    
    hipMemcpy(dev_results, results, sizeof(Test_Result) * num_results, hipMemcpyHostToDevice);
    
    int done = 0;
    while(done < num_results)
    {        
         //Run Tests
        Miller_Rabin_Kernal<<<dimGrid, dimBlock>>>(dev_results + done, dev_rand_state);
        
        done += KERNEL_SIZE;
    }
   
    //Transfer results back to cpu
    hipMemcpy(results, dev_results, sizeof(Test_Result) * num_results, hipMemcpyDeviceToHost);
            
    //Clean up memory
    hipFree(dev_results);
    hipFree(dev_rand_state);
}

uint32_t serial_modular_exponent_32(uint32_t base, uint32_t power, uint32_t modulus) 
{
    uint64_t result = 1;
    
    while(power > 0)
    {
        if((power & 1) == 1)
        {
            result = (result * base) % modulus;
        }
        
        power >>= 1;
        base = (base * base) % modulus;
    }
    
    return (uint32_t) result; /* Will not truncate since modulus is a uint32_t */
}

void Miller_Rabin_Serial(Test_Result *results, int num_results)
{
    printf("Running Serial on %d numbers\n", num_results);
    for(int index = 0; index < num_results; index++)
    {
        uint32_t test_num = results[index].num;
        for(int j = 0; j < 32; j++)
        {
            //mod random number so that a < n
            uint32_t a = 0;
            while(a < 1 || a > test_num - 1)
                a = rand() % test_num;

            //do test
            uint32_t a_to_power, s, d, i;

            //16-bit compute s and d
            s = 0;
            d = test_num - 1;

            while ((d % 2) == 0) 
            {
                d >>= 1;
                s++;
            }
            
            if(s == 0) //Even number so cannot be prime
            {
                results[index].passed = 1;
                continue;
            }

            a_to_power = serial_modular_exponent_32(a, d, test_num);
            
            if (a_to_power == 1)
            {
                //printf("Thread #%d %u Return 1\n", j, test_num);
                continue;
            }

            for(i = 0; i < s - 1; i++) 
            {
                if (a_to_power == test_num - 1)
                {
                    //printf("Thread #%d %u Return 2\n", j, test_num);
                    continue;
                }

                a_to_power = serial_modular_exponent_32(a_to_power, 2, test_num);
            }

            if (a_to_power == test_num - 1)
            {
                //printf("Thread #%d %u Return 3\n", j, test_num);
                continue;
            }
            
            //printf("Thread #%d %u Return NOT\n", j, test_num);
            results[index].passed = 1;
        }      
    }
}

int main(int argc, char **argv)
{
    Test_Result *results;
    int num_results = atoi(argv[1]);        
    int runCUDA = 1;
     
    results = (Test_Result *) malloc(sizeof(Test_Result) * num_results);

    //Generate or get from a file the test numbers NOTE dont test 2
    for(uint32_t i = 0; i < num_results; i++)
    {    
        results[i].num = i + 2;
        results[i].passed = 0;
    }
    
    if(argc >= 3)
    {
        if(strcmp(argv[2], "-s") == 0)
            runCUDA = 0; 
    }
    
    if(runCUDA)
        run_kernel(results, num_results);
    else
        Miller_Rabin_Serial(results, num_results);

    //Print results
    for(int i = 0; i < num_results; i++)
    {  
        //printf("%d %d\n", results[i].num, results[i].passed);      
        if(results[i].passed == PASSED)
            printf("%d\n", results[i].num);      
    }
    
    printf("Tested %d numbers\n", num_results);
    free(results);
}

